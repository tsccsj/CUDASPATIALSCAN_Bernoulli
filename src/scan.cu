#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define BLOCKSIZE 16
#define NumRadius 100
#define RadiusIncrement 10000

__global__ void scanKernel(float * dX, float * dY, int * dI, int nPoints, int * dWC, int * dWP, float * dLike, int nCol, int nRow, float xMin, float yMax, float cellSize, int nCase)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idInThread = threadIdx.y * blockDim.x + threadIdx.x;
	
	float cellX = xMin + cellSize * x;
	float cellY = yMax - cellSize * y;

	float dist;

	__shared__ float sX[BLOCKSIZE * BLOCKSIZE];
	__shared__ float sY[BLOCKSIZE * BLOCKSIZE];
	__shared__ int sI[BLOCKSIZE * BLOCKSIZE];

	int wCCell[NumRadius];
	int wPCell[NumRadius];

	for(int i = 0; i < NumRadius; i++)
	{
		wCCell[i] = 0;
		wPCell[i] = 0;
	}
	
	int pointProcessed;
	int pointToProcess = BLOCKSIZE * BLOCKSIZE;

	for(pointProcessed = 0; pointProcessed < nPoints; pointProcessed += BLOCKSIZE * BLOCKSIZE)
	{
		if(pointProcessed + pointToProcess > nPoints)
		{
			pointToProcess = nPoints - pointProcessed;
		}

		if(idInThread < pointToProcess)
		{
			sX[idInThread] = dX[pointProcessed + idInThread];
			sY[idInThread] = dY[pointProcessed + idInThread];
			sI[idInThread] = dI[pointProcessed + idInThread];
		}

		__syncthreads();
		for(int i = 0; i < pointToProcess; i++)
		{
			dist = sqrt((cellX - sX[i]) * (cellX - sX[i]) + (cellY - sY[i]) * (cellY - sY[i]));
			if(sI[i] > 0)
			{
				for(int j = dist / RadiusIncrement; j < NumRadius; j++)
				{
					wCCell[j] ++;
					wPCell[j] ++;
				}
			}
			else
			{
				for(int j = dist / RadiusIncrement; j < NumRadius; j++)
				{
					wPCell[j] ++;
				}
			}
		}
		__syncthreads();
	}

	int cellID = (y * nCol + x) * NumRadius;
	if(x < nCol && y < nRow)
	{
		for(int i = 0; i < NumRadius; i++)
		{
			//If it is a cold spot, ignore it (it doesn't matter to do this)
			if(wCCell[i] * nPoints < wPCell[i] * nCase)
				wCCell[i] = 0;

			dWC[cellID + i] = wCCell[i];
			dWP[cellID + i] = wPCell[i];

			dLike[cellID + i] = wCCell[i] * log((float)wCCell[i]/wPCell[i]) + (wPCell[i]-wCCell[i]) * log((float)(wPCell[i]-wCCell[i])/wPCell[i]) + (nCase-wCCell[i]) * log((float)(nCase-wCCell[i])/(nPoints-wPCell[i])) + (nPoints-wPCell[i]-nCase+wCCell[i]) * log((float)(nPoints-wPCell[i]-nCase+wCCell[i])/(nPoints-wPCell[i])); 
/*		
			if(wCCell[i] > wPCell[i])
			{
				dLike[cellID + i] = 100.0;
			}
*/
		}
	}

}

__global__ void scanKernelMC(float * dX, float * dY, int * dI, int nPoints, int * dWC, int * dAbove, int nCol, int nRow, float xMin, float yMax, int cellSize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idInThread = threadIdx.y * blockDim.x + threadIdx.x;
	
	float cellX = xMin + cellSize * (x + 0.5);
	float cellY = yMax - cellSize * (y + 0.5);

	float dist;

	__shared__ float sX[BLOCKSIZE * BLOCKSIZE];
	__shared__ float sY[BLOCKSIZE * BLOCKSIZE];
	__shared__ int sI[BLOCKSIZE * BLOCKSIZE];

	int wCSim[NumRadius];

	for(int i = 0; i < NumRadius; i++)
	{
		wCSim[i] = 0;
	}
	
	int pointProcessed;
	int pointToProcess = BLOCKSIZE * BLOCKSIZE;

	for(pointProcessed = 0; pointProcessed < nPoints; pointProcessed += BLOCKSIZE * BLOCKSIZE)
	{
		if(pointProcessed + pointToProcess > nPoints)
		{
			pointToProcess = nPoints - pointProcessed;
		}

		if(idInThread < pointToProcess)
		{
			sX[idInThread] = dX[pointProcessed + idInThread];
			sY[idInThread] = dY[pointProcessed + idInThread];
			sI[idInThread] = dI[pointProcessed + idInThread];
		}

		__syncthreads();
		for(int i = 0; i < pointToProcess; i++)
		{
			dist = sqrt((cellX - sX[i]) * (cellX - sX[i]) + (cellY - sY[i]) * (cellY - sY[i]));
			if(sI[i] > 0)
			{
				for(int j = dist / RadiusIncrement; j < NumRadius; j++)
				{
					wCSim[j] ++;
				}
			}
		}
		__syncthreads();
	}

	int cellID = (y * nCol + x) * NumRadius;
	if(x < nCol && y < nRow)
	{
		for(int i = 0; i < NumRadius; i++)
		{
			if(dWC[cellID + i] < wCSim[i])
			{
				dAbove[cellID + i] ++;
			}
		}
	}
}


void createSample(int * ind, int nPoints, int nCase)
{
	int chosen;
	for(int i = 0; i < nPoints; i++)
		ind[i] = 0;
	while(nCase > 0)
	{
		chosen = (double)rand()/RAND_MAX * nPoints;
		if(ind[chosen] == 0)
		{
			nCase --;
			ind[chosen] = 1;
		}
	}
}

void cacScan(float * xCor, float * yCor, int * ind, int nPoints, int * wCase, int * wPop, float * like, float * pValue, int nCol, int nRow, float xMin, float yMax, float cellSize, int numSim, int nCase)
{
	float * dX;
	float * dY;
	int * dI;
	int * dWC;
	int * dWP;
	float * dLike;


	hipError_t err;

	err = hipMalloc((void **) &dX, nPoints * sizeof(float));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **) &dY, nPoints * sizeof(float));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **) &dI, nPoints * sizeof(int));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **) &dWC, nRow * nCol * NumRadius * sizeof(int));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **) &dWP, nRow * nCol * NumRadius * sizeof(int));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **) &dLike, nRow * nCol * NumRadius * sizeof(float));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	err = hipMemcpy(dX, xCor, sizeof(float) * nPoints, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(dY, yCor, sizeof(float) * nPoints, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(dI, ind, sizeof(int) * nPoints, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}


//Kernel Goes Here
	dim3 dimBlock (BLOCKSIZE, BLOCKSIZE, 1);
	int gridX = int(ceil((float)nCol / BLOCKSIZE));
	int gridY = int(ceil((float)nRow / BLOCKSIZE));
	dim3 dimGrid (gridX, gridY, 1);

	scanKernel<<<dimGrid, dimBlock>>>(dX, dY, dI, nPoints, dWC, dWP, dLike, nCol, nRow, xMin, yMax, cellSize, nCase);

	err = hipDeviceSynchronize();
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	err = hipMemcpy(wCase, dWC, sizeof(int) * nCol * nRow * NumRadius, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(wPop, dWP, sizeof(int) * nCol * nRow * NumRadius, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(like, dLike, sizeof(float) * nCol * nRow * NumRadius, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}


	err = hipFree(dWP);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipFree(dLike);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

/*Test
	
	for(int i = 0; i < nCol * nRow * NumRadius; i++)
	{
		if(wCase[i] > wPop[i])
		{
			printf("ID: %d\twCase= %d\twPop= %d\n", i, wCase[i], wPop[i]);
		}
	}
*/

	//For simulation and pValue
	int * numAbove;
	if(NULL == (numAbove = (int *) malloc(sizeof(int) * nCol * nRow * NumRadius)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	for(int i = 0; i < nCol * nRow * NumRadius; i ++)
	{
		numAbove[i] = 0;
	}

	int * dAbove;
	err = hipMalloc((void **) &dAbove, nCol * nRow * NumRadius * sizeof(int));
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(dAbove, numAbove, sizeof(int) * nCol * nRow * NumRadius, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	time_t t;
	srand((unsigned) time(&t));

	//Loop of simulation
	for(int i = 0; i < numSim; i++)
	{
		printf("Begin simulation %d\n", i);
		createSample(ind, nPoints, nCase);
		err = hipMemcpy(dI, ind, sizeof(int) * nPoints, hipMemcpyHostToDevice);
		if(err != hipSuccess)
		{
			printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
			exit(1);
		}

		//Kernel Goes Here

		scanKernelMC<<<dimGrid, dimBlock>>>(dX, dY, dI, nPoints, dWC, dAbove, nCol, nRow, xMin, yMax, cellSize);
		
	}

	err = hipMemcpy(numAbove, dAbove, sizeof(int) * nCol * nRow * NumRadius, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	err = hipFree(dWC);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipFree(dX);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipFree(dY);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipFree(dI);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipFree(dAbove);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	for(int i = 0; i < nCol * nRow * NumRadius; i ++)
	{
		pValue[i] = (float)(numAbove[i] + 1) / (numSim + 1);
	}

	free(numAbove);
}
