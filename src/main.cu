#include "hip/hip_runtime.h"
#include <stdio.h>
#include "io.cuh"
#include "scan.cuh"

#define NumRadius 100
#define RadiusIncrement 10000

int main(int argc, char ** argv)
{
	FILE * file;

	float * xCor;
	float * yCor;
	int * ind;

	int nPoints, nCase;
	
	float xMin = -2380000, xMax = 2280000, yMin = -1520000, yMax = 1420000;
	float cellSize = 10000;

	int nRow = ceil((yMax - yMin)/cellSize);
	int nCol = ceil((xMax - xMin)/cellSize);
	yMax = yMin + nRow * cellSize;
	xMax = xMin + nCol * cellSize;

	nRow ++;
	nCol ++;

	printf("xMax = %f\txMin = %f\tyMax = %f\tyMin = %f\n", xMax, xMin, yMax, yMin);
	printf("nRow = %d\tnCol = %d\n",nRow,nCol);

	if(NULL == (file = fopen("/home/ygao29/cudaScan/data/2014_01_01", "r")))
	{
		printf("ERROR: Cannot open your flu points file \n");
		exit(1);
	}

	nPoints = getNumPoints(file);


	if(NULL == (xCor = (float *) malloc(sizeof(float)*nPoints)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (yCor = (float *) malloc(sizeof(float)*nPoints)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (ind = (int *) malloc(sizeof(int)*nPoints)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}


	readFile(file, xCor, yCor, ind, nPoints, nCase);

	printf("nCase = %d\tnPoints = %d\n",nCase,nPoints);
	
	fclose(file);


	int * wCase;
	int * wPop;
	float * like;
	float * pValue;

	if(NULL == (wCase = (int *) malloc(sizeof(int) * nCol * nRow * NumRadius)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (wPop = (int *) malloc(sizeof(int) * nCol * nRow * NumRadius)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (like = (float *) malloc(sizeof(float) * nCol * nRow * NumRadius)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (pValue = (float *) malloc(sizeof(float) * nCol * nRow * NumRadius)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}


	cacScan(xCor, yCor, ind, nPoints, wCase, wPop, like, pValue, nCol, nRow, xMin, yMax, cellSize, 99, nCase);

	free(xCor);
	free(yCor);
	free(ind);

/*
	if(NULL == (file = fopen("/home/ygao29/cudaScan/result/2014_01_01", "w")))
	{
		printf("ERROR: Cannot open your flu points file \n");
		exit(1);
	}

	for(int i = 0; i < nCol * nRow; i++)
	{
		fprintf(file, "%d:%d:%f:%f\t%d:%d:%f:%f\t%d:%d:%f:%f\t%d:%d:%f:%f\t%d:%d:%f:%f\n", wCase[i * NumRadius], wPop[i * NumRadius], like[i * NumRadius], pValue[i * NumRadius], wCase[i * NumRadius + 1], wPop[i * NumRadius + 1], like[i * NumRadius + 1], pValue[i * NumRadius + 1], wCase[i * NumRadius + 2], wPop[i * NumRadius + 2], like[i * NumRadius + 2], pValue[i * NumRadius + 2], wCase[i * NumRadius + 3], wPop[i * NumRadius + 3], like[i * NumRadius + 3], pValue[i * NumRadius + 3], wCase[i * NumRadius + 4], wPop[i * NumRadius + 4], like[i * NumRadius + 4], pValue[i * NumRadius + 4]);
	}

	fclose(file);
*/	

	int xID, yID, radiusID, ID;
	int nClusters = 50;
	float shieldFloat;
	int shieldInt;

	for(int i = 0; i < nClusters; i++)
	{
		ID = -1;
		for(int j = 0; j < nCol * nRow * NumRadius; j++)
		{
			if(wCase[j] != 0)
			{
				if(ID == -1 || like[ID] < like[j])
					ID = j;
			}
		}

		radiusID = ID % NumRadius;
		xID = (ID / NumRadius) % nCol;
		yID = ID / NumRadius / nCol;


		printf("#####################\n");
		printf("Cluster %d\n", i);
		printf("x: %f   y: %f   radius: %f\n", xMin + cellSize * xID, yMax - cellSize * yID, (float)(radiusID + 1) * RadiusIncrement);
		printf("NumPopulation: %d   NumCase: %d\n", wPop[ID], wCase[ID]);
		printf("Expected number of cases: %f\n", (float)wPop[ID] * nCase / nPoints);
		printf("Log likelihood: %f   P-Value: %f\n\n", like[ID], pValue[ID]);

		//A cluster can't be in another's center
		shieldFloat = (radiusID + 1) * RadiusIncrement / cellSize;
 		shieldInt = (int)shieldFloat;

		shieldFloat = shieldFloat * shieldFloat;

		for(int j = xID - shieldInt; j < xID + shieldInt; j++)
		{
			if(j < 0)
				j = 0;
			else if(j >= nCol)
				break;

			for(int k = yID - shieldInt; k < yID + shieldInt; k++)
			{
				if(k < 0)
					k = 0;
				else if (k >= nRow)
					break;
				if((k - yID) * (k - yID) + (j - xID) * (j - xID) < shieldFloat)
				{
					for(int l = 0; l < NumRadius; l ++)
					{
						wCase[l + NumRadius * j + NumRadius * nCol * k] = 0;
					}
				}
			}
		}
		


	}

	free(wCase);
	free(wPop);
	free(like);
	free(pValue);

	return 0;

}
